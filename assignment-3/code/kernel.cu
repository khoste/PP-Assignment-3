
#include <hip/hip_runtime.h>
// index trap as a 2D-array 
__device__ inline float idx(float *trap, int x, int y, int w) {
	return trap[(y*w)+x];
} 

// compute pixel value
__device__ inline float compute_pixel(float *trap, float omega, int x, int y, int w) {
	return (omega / 4.0) * (idx(trap, x-1, y  , w)
						 +  idx(trap, x+1, y  , w)
						 +  idx(trap, x  , y-1, w)
						 +  idx(trap, x  , y+1, w))
						 + (1.0 - omega) * idx(trap, x,y,w);
}

/*
 * simple
 */
__global__ void simple(float *trap, int h, int w, float omega, float epsilon, int iter, float *delta) {
	// implement me
  printf ("hi from simple ");
  
	
}

/*
 * rbshi 
 */
__global__ void rb(float *trap, int h, int w, float omega, float epsilon, int iter, float *delta) {
	// implement me
	 printf ("hi from rb ");
}

/*
 * dbuf
 */
__global__ void dbuf(float *trap, int h, int w, float omega, float epsilon, int iter, float *delta) {
	//implement me
	//printf ("hi from dbuf ");
}
