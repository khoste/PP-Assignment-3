#include "hip/hip_runtime.h"
// index trap as a 2D-array 
__device__ inline float idx(float *trap, int x, int y, int w) {
	return trap[(y*w)+x];
} 

// compute pixel value
__device__ inline float compute_pixel(float *trap, float omega, int x, int y, int w) {
	return (omega / 4.0) * (idx(trap, x-1, y  , w)
						 +  idx(trap, x+1, y  , w)
						 +  idx(trap, x  , y-1, w)
						 +  idx(trap, x  , y+1, w))
						 + (1.0 - omega) * idx(trap, x,y,w);
}

/*
 * simple
 */
__global__ void simple(float *trap, int h, int w, float omega, float epsilon, int iter, float *delta) {
	// implement me
   	const int t_ID = threadIdx.x + blockIdx.y * blockDim.x;

  printf ("hi from simple ");
  if (threadIdx.x == 4 && blockIdx.y == 3){
  	// printf("t_Id=",t_ID);
  int py = threadIdx.x;
  int px = blockIdx.y;
  // if(blockIdx.y == 5 && threadIdx.x == 3){
  	 printf("threadIdx.x=%d, blockIdx.y=%d, blockDim.x=%d, t_ID=%d, px=%d, py=%d\n",threadIdx.x, blockIdx.y, blockDim.x, t_ID, px, py);
         printf ("This is from colonel");  	
	 trap1[t_ID] = compute_pixel(trap, omega, py, px, w);
  	// printf("trap[%d]=%f\n", t_ID, trap[t_ID]);
  	delta=fabs(trap1[t_ID]-trap[t_ID]);
	
}

/*
 * rbshi 
 */
__global__ void rb(float *trap, int h, int w, float omega, float epsilon, int iter, float *delta) {
	// implement me
	 printf ("hi from rb ");
}

/*
 * dbuf
 */
__global__ void dbuf(float *trap, int h, int w, float omega, float epsilon, int iter, float *delta) {
	//implement me
	//printf ("hi from dbuf ");
}
