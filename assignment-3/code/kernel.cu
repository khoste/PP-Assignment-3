#include "hip/hip_runtime.h"
// index trap as a 2D-array 
__device__ inline float idx(float *trap, int x, int y, int w) {
	return trap[(y*w)+x];
} 

// compute pixel value
__device__ inline float compute_pixel(float *trap, float omega, int x, int y, int w) {
	return (omega / 4.0) * (idx(trap, x-1, y  , w)
						 +  idx(trap, x+1, y  , w)
						 +  idx(trap, x  , y-1, w)
						 +  idx(trap, x  , y+1, w))
						 + (1.0 - omega) * idx(trap, x,y,w);
}

/*
 * simple
 */
__global__ void simple(float *trap, int h, int w, float omega, float epsilon, int iter, float *delta) {
	// implement me
  
int i = 0,x,y;
printf('from simple');
while(i < iter)
 {
     	 
        for (y = 1; y < h - 1; y++) 
           {  
              for (x = 1; x < w - 1; x++) {
                   float old = trap[x][y];
                   float new = compute_pixel(trap, omega, x, y,  w);
                   trap[x][y] = new;
                   delta += fabs(old - new);
		   printf('from simple delta %f',delta);
                   }
             }
     } 
 
}

/*
 * rbshi 
 */
__global__ void rb(float *trap, int h, int w, float omega, float epsilon, int iter, float *delta) {
	// implement me
	 printf ("hi from rb ");
}

/*
 * dbuf
 */
__global__ void dbuf(float *trap, int h, int w, float omega, float epsilon, int iter, float *delta) {
	//implement me
	//printf ("hi from dbuf ");
}
